
__host__ void dummy_input(int *input, int size)
{
  for(int i=0; i<size; i++)
    input[i]=1;
}

int  main(int argc, char** argv)
{
  int *d_input, *d_output;
  int *input, *partial_result;
  int width,height,size;


  dim3 block,grid;


  width   = 1024;
  height  = 1024;
  if (argc>2)
  {
    width   = atoi(argv[1]);
    height  = atoi(argv[2]);
  }
  size = width*height;
 
  block.x = TBLOCK_SIZE;
  grid.x  = ( ( size + block.x - 1 ) / ( block.x * DB_PER_TB ) );


  input          = (int *)malloc(size  *sizeof(int));
  partial_result = (int *)malloc(grid.x*sizeof(int));
  hipMalloc( (void **)&d_input,     size*sizeof(int));
  hipMalloc( (void **)&d_output,    grid.x*sizeof(int));

  dummy_input(input,size);  

  hipMemcpy(d_input, input, size*sizeof(int), hipMemcpyHostToDevice);
  reduce<TBLOCK_SIZE><<<grid, block>>>(d_input, d_output, size);
  hipMemcpy(partial_result, d_output, grid.x*sizeof(int), hipMemcpyDeviceToHost);


  int result = 0;
  for (int i=0; i<grid.x; i++)
    result += partial_result[i];


  return EXIT_SUCCESS;
}